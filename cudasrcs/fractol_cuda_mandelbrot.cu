#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   fractol_cuda_mandelbrot.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/17 10:41:44 by aanzieu           #+#    #+#             */
/*   Updated: 2017/03/17 11:20:31 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../cudainclude/cuda_call.h"

__global__ void mandelbrot_2(unsigned int* a_d, int constw, int consth, double xmin, double xmax, double ymin, double ymax, double zoom, float offx, float offy, float max, int power)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = y * constw + x;
	if(y >= constw || x >= consth || index >= N) return;
	float cr = (((double)x / constw) * (xmax - xmin)) * zoom + xmin + offx;
	float ci = (((double)y / consth) * (ymax - ymin)) * zoom + ymin + offy;
	float zr = 0;
	float zi = 0;
	float tmpr;
	float tmpi;
	int		i = 0;
	while (zr * zr + zi * zi < 4 && i < max)
	{
		if (power == 3)
		{
			tmpr = pow(zr, 3) - 3 * zr * zi * zi + cr;
			tmpi = 3 * zr * zr * zi - pow(zi, 3) + ci;
		}
		if (power == 4)
		{
		tmpr = pow(zr, 4) - 6 * pow(zr, 2) * pow(zi, 2) + pow(zi, 4) + cr;
		tmpi = 4 * pow(zr, 3) * zi - 4 * zr * pow(zi, 3) + ci;
		}
		if (power == 2)
		{
			tmpr = zr * zr - zi * zi + cr;
			tmpi = zr * zi * 2 + ci;
		}
		zr = tmpr;
		zi = tmpi;
		i++;
	}
	a_d[index] = i;
}

extern "C" void cuda_call_mandelbrot(unsigned int* a_h, unsigned int size, t_env *e, int end)
{
	unsigned int *a_d = NULL;
	static dim3 blockDim(16, 16);
	static dim3 gridDim(WIN_WIDTH / blockDim.x, WIN_HEIGTH / blockDim.y);

	e->view.xmin = -2;
	e->view.xmax = 0.5;
	e->view.ymin = -1.25;
	e->view.ymax = 1.25;
	hipMalloc((void**) &a_d, size*sizeof(unsigned int));
	mandelbrot_2 <<< gridDim, blockDim >>> (a_d, WIN_WIDTH, WIN_HEIGTH, e->view.xmin, e->view.xmax, e->view.ymin, e->view.ymax, e->plan.zoom, e->plan.offx, e->plan.offy, e->plan.i_max, e->plan.power);
	hipMemcpy(a_h, a_d, size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(a_d);
	if(end == 1)
		hipDeviceReset();
}
