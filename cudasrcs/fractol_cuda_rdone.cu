#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   fractol_cuda_rdone.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/17 10:41:44 by aanzieu           #+#    #+#             */
/*   Updated: 2017/03/17 11:20:31 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../cudainclude/cuda_call.h"

__global__ void rdone_2(unsigned int* a_d, int constw, int consth, double xmin, double xmax, double ymin, double ymax, double zoom, float offx, float offy, float cr, float ci, float max, int power)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = y * constw + x;
	if(y >= constw || x >= consth || index >= N) return;
	x = (((double)x / constw) * (xmax - xmin)) * zoom + xmin + offx;
	y = (((double)y / consth) * (ymax - ymin)) * zoom + ymin + offy;
	x = abs(x);
	y = abs(y);
	int i = 0;
	while ((x >0 || y > 0) && i < max)
	{
		if (x % 3 == 1 && y % 3 == 1)
		{
			a_d[index] = 0;
			return;
		}
		x /= 3;
		y /= 3;
		i++;
	}
	a_d[index] = i;
}
extern "C" void cuda_call_rdone(unsigned int* a_h, unsigned int size, t_env *e, int end)
{
	unsigned int *a_d = NULL;
	static dim3 blockDim(16, 16);
	static dim3 gridDim(WIN_WIDTH / blockDim.x, WIN_HEIGTH / blockDim.y);

	e->view.xmin = -2.0f;
	e->view.xmax = 2.0f;
	e->view.ymin = -2.0f;
	e->view.ymax = 2.0f;
	hipMalloc((void**) &a_d, size*sizeof(unsigned int));
	rdone_2 <<< gridDim, blockDim >>> (a_d, WIN_WIDTH, WIN_HEIGTH, e->view.xmin, e->view.xmax, e->view.ymin, e->view.ymax, e->plan.zoom, e->plan.offx, e->plan.offy, e->plan.cr, e->plan.ci, e->plan.i_max, e->plan.power);
	hipMemcpy(a_h, a_d, size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(a_d);
	if(end == 1)
		hipDeviceReset();
}
