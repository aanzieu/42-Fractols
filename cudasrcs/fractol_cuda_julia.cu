#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   fractol_cuda_julia.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/17 10:41:44 by aanzieu           #+#    #+#             */
/*   Updated: 2017/03/17 11:20:31 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../cudainclude/cuda_call.h"

__global__ void julia_2(unsigned int* a_d, int constw, int consth, double xmin, double xmax, double ymin, double ymax, double zoom, float offx, float offy, float cr, float ci, float max, int power)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = y * constw + x;
	if(y >= constw || x >= consth || index >= N) return;
	float zr = (((double)x / constw) * (xmax - xmin)) * zoom + xmin + offx;
	float zi = (((double)y / consth) * (ymax - ymin)) * zoom + ymin + offy;
	float tmpr;
	float tmpi;
	int		i = 0;
	
	while (zr * zr + zi * zi < 4  && i < max)
	{
		tmpr = zr * zr - zi * zi + cr;
		tmpi = zr * zi * power + ci;
		zr = tmpr;
		zi = tmpi;
		i++;
	}
	a_d[index] = i;
}
extern "C" void cuda_call_julia(unsigned int* a_h, unsigned int size, t_env *e, int end)
{
	unsigned int *a_d = NULL;
	static dim3 blockDim(16, 16);
	static dim3 gridDim(WIN_WIDTH / blockDim.x, WIN_HEIGTH / blockDim.y);
	
	e->view.xmin = -2;
	e->view.xmax = 2;
	e->view.ymin = -2;
	e->view.ymax = 2;
	hipMalloc((void**) &a_d, size*sizeof(unsigned int));
	julia_2 <<< gridDim, blockDim >>> (a_d, WIN_WIDTH, WIN_HEIGTH, e->view.xmin, e->view.xmax, e->view.ymin, e->view.ymax, e->plan.zoom, e->plan.offx, e->plan.offy, e->plan.cr, e->plan.ci, e->plan.i_max, e->plan.power);
	hipMemcpy(a_h, a_d, size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(a_d);
	if(end == 1)
		hipDeviceReset();
}
